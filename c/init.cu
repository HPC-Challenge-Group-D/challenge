#include "hip/hip_runtime.h"
//
// Created by Patrik Rac on 08.03.23.
//

#include "proc_info.h"
#include "init.h"

__global__
void initDataKernel(double *v, double *f, int nx, int ny, int offset_x, int offset_y)
{
    int ixx = threadIdx.x + blockIdx.x * blockDim.x;
    int sx = blockDim.x * gridDim.x;

    int iyy = threadIdx.y + blockIdx.y * blockDim.y;
    int sy = blockDim.y * gridDim.y;

    // Initialise input
    for (int iy = iyy; iy < ny; iy+=sy)
        for (int ix = ixx; ix < nx; ix+=sx)
        {
            v[nx*iy+ix] = 0.0;

            const double x = 2.0 * (ix+offset_x) / (NX - 1.0) - 1.0;
            const double y = 2.0 * (iy+offset_y) / (NY - 1.0) - 1.0;
            f[nx*iy+ix] = sin(x + y);
        }
}

void initDevice()
{
    /*Set the appropriate device before the call to MPI_init()*/
    char * localRankStr = NULL;
    int rank = 0, deviceCount = 0;

    // We extract the local rank initialization using an environment variable
    if ((localRankStr = getenv("OMPI_COMM_WORLD_LOCAL_RANK")) != NULL)
    {
        rank = atoi(localRankStr);
    }
    hipGetDeviceCount(&deviceCount);
    hipSetDevice(rank % deviceCount);
}

/*
 * Prepares the memory of v, vp and f on the device and initializes v and f with the correct values
 */
void prepareDataMemory(double *v, double *vp, double *f, int nx, int ny, int offset_x, int offset_y)
{
    int deviceId;
    hipGetDevice(&deviceId);
    dim3 threadsPerBlock = dim3(16, 16);
    dim3 numberOfBlocks = dim3(8,8);
    hipMallocManaged(&v, nx * ny * sizeof(double));
    hipMallocManaged(&f, nx * ny * sizeof(double));

    hipMallocManaged(&vp, nx * ny * sizeof(double));

    /*Move the data used for the computation to the device*/
    hipMemAdvise(v, nx*ny*sizeof(double), (hipMemoryAdvise) 2, deviceId);
    hipMemAdvise(vp, nx*ny*sizeof(double), (hipMemoryAdvise) 2, deviceId);
    hipMemAdvise(f, nx*ny*sizeof(double), (hipMemoryAdvise) 1, deviceId);

    hipMemPrefetchAsync(v, nx*ny*sizeof(double), deviceId);
    hipMemPrefetchAsync(vp, nx*ny*sizeof(double), deviceId);
    hipMemPrefetchAsync(f, nx*ny*sizeof(double), deviceId);

    initDataKernel<<<numberOfBlocks, threadsPerBlock>>>(v,f,nx,ny,offset_x, offset_y);
}

unsigned int prepareMiscMemory(double *w, double *e, double *w_device, double *e_device)
{
    int deviceId;
    hipGetDevice(&deviceId);
    dim3 threadsPerBlock = dim3(16, 16);
    dim3 numberOfBlocks = dim3(8,8);
    unsigned int num_gpu_threads = (numberOfBlocks.x * threadsPerBlock.x) * (numberOfBlocks.y * threadsPerBlock.y);
    hipMallocManaged(&w_device, num_gpu_threads*sizeof(double));
    hipMallocManaged(&e_device, num_gpu_threads*sizeof(double));

    hipMemPrefetchAsync(w_device, num_gpu_threads*sizeof(double), deviceId);
    hipMemPrefetchAsync(e_device, num_gpu_threads*sizeof(double), deviceId);

    hipMallocManaged(&w, sizeof(double));
    hipMallocManaged(&e, sizeof(double));
    return num_gpu_threads;
}

void freeDataMemory(double *v, double *vp, double *f)
{
    hipFree(v);
    hipFree(vp);
    hipFree(f);
}


void freeMiscMemory(double *w, double *e, double *w_device, double *e_device)
{
    hipFree(w);
    hipFree(e);
    hipFree(w_device);
    hipFree(e_device);
}