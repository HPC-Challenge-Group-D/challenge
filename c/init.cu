#include "hip/hip_runtime.h"
//
// Created by Patrik Rac on 08.03.23.
//

#include "init.h"

__global__
void initDataKernel(double *v, double *f, int nx, int ny, int offset_x, int offset_y)
{
    int ixx = threadIdx.x + blockIdx.x * blockDim.x;
    int sx = blockDim.x * gridDim.x;

    int iyy = threadIdx.y + blockIdx.y * blockDim.y;
    int sy = blockDim.y * gridDim.y;

    // Initialise input
    for (int iy = iyy; iy < ny; iy+=sy)
        for (int ix = ixx; ix < nx; ix+=sx)
        {
            v[nx*iy+ix] = 0.0;

            const double x = 2.0 * (ix+offset_x) / (NX - 1.0) - 1.0;
            const double y = 2.0 * (iy+offset_y) / (NY - 1.0) - 1.0;
            f[nx*iy+ix] = sin(x + y);
        }
}

extern "C" void initDevice()
{
    /*Set the appropriate device before the call to MPI_init()*/
    char * localRankStr = NULL;
    int rank = 0;
    //int deviceCount = 0;

    // We extract the local rank initialization using an environment variable
    if ((localRankStr = getenv("SLURM_LOCALID")) != NULL)
    {
        rank = atoi(localRankStr);
    }
    else
    {
        printf("Could not determine the appropriate local rank!\n");
    }
    //hipGetDeviceCount(&deviceCount);
    //printf("There are %d devices\n", deviceCount);
    hipSetDevice(rank);
}

/*
 * Prepares the memory of v, vp and f on the device and initializes v and f with the correct values
 */
extern "C" void prepareDataMemory(double **v, double **vp, double **f, int nx, int ny, int offset_x, int offset_y)
{
    int deviceId;
    hipGetDevice(&deviceId);
    dim3 threadsPerBlock = dim3(16, 16);
    dim3 numberOfBlocks = dim3(8,8);
    hipMallocManaged(v, nx * ny * sizeof(double));
    hipMallocManaged(f, nx * ny * sizeof(double));

    hipMallocManaged(vp, nx * ny * sizeof(double));

    /*Move the data used for the computation to the device*/
    hipMemAdvise(*v, nx*ny*sizeof(double), (hipMemoryAdvise) 2, deviceId);
    hipMemAdvise(*vp, nx*ny*sizeof(double), (hipMemoryAdvise) 2, deviceId);
    hipMemAdvise(*f, nx*ny*sizeof(double), (hipMemoryAdvise) 1, deviceId);

    hipMemPrefetchAsync(*v, nx*ny*sizeof(double), deviceId);
    hipMemPrefetchAsync(*vp, nx*ny*sizeof(double), deviceId);
    hipMemPrefetchAsync(*f, nx*ny*sizeof(double), deviceId);

    initDataKernel<<<numberOfBlocks, threadsPerBlock>>>(*v,*f,nx,ny,offset_x, offset_y);
}

extern "C" unsigned int prepareMiscMemory(double **w, double **e, double **w_device, double **e_device)
{
    int deviceId;
    hipGetDevice(&deviceId);
    dim3 threadsPerBlock = dim3(16, 16);
    dim3 numberOfBlocks = dim3(8,8);
    unsigned int num_gpu_threads = (numberOfBlocks.x * threadsPerBlock.x) * (numberOfBlocks.y * threadsPerBlock.y);
    hipMallocManaged(w_device, num_gpu_threads*sizeof(double));
    hipMallocManaged(e_device, num_gpu_threads*sizeof(double));

    hipMemPrefetchAsync(*w_device, num_gpu_threads*sizeof(double), deviceId);
    hipMemPrefetchAsync(*e_device, num_gpu_threads*sizeof(double), deviceId);

    hipMallocManaged(w, sizeof(double));
    hipMallocManaged(e, sizeof(double));
    return num_gpu_threads;
}

extern "C" void freeDataMemory(double **v, double **vp, double **f)
{
    hipFree(*v);
    hipFree(*vp);
    hipFree(*f);
}


extern "C" void freeMiscMemory(double **w, double **e, double **w_device, double **e_device)
{
    hipFree(*w);
    hipFree(*e);
    hipFree(*w_device);
    hipFree(*e_device);
}