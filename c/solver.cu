#include "hip/hip_runtime.h"
//
// Created by Patrik Rac on 11.03.23.
//

/*
 * Copyright (c) 2021, Dirk Pleiter, KTH
 *
 * This source code is in parts based on code from Jiri Kraus (NVIDIA) and
 * Andreas Herten (Forschungszentrum Juelich)
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND Any
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR Any DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON Any THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN Any WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "proc_info.h"

#include <cub/device/device_reduce.cuh>

/*Get headers of the device reduction functions*/
void deviceReduce(double *in, double* out, int N);
void deviceReduceMax(double *in, double* out, int N);

/*
 * Kernel to perform the Jacobi Step
 * Results of the steps are then rewritten into the original array
 * The Device calculates its own error and weight value at position tid
 */
__global__
void jacobiStepKernel(double *vp, double *v, double *f, int nx, int ny, double *e, double *w)
{
    int ixx = threadIdx.x + blockIdx.x * blockDim.x;
    int sx = blockDim.x * gridDim.x;

    int iyy = threadIdx.y + blockIdx.y * blockDim.y;
    int sy = blockDim.y * gridDim.y;

    /*Thread id for computation of the local weights and errors*/
    const unsigned int tid = ixx*sy + iyy;

    e[tid] = 0.0;
    w[tid] = 0.0;


    if(ixx < 1)
        ixx+=sx;

    if(iyy < 1)
        iyy+=sy;

    /*
     * Perform actual Jacobi Step
     */
    for (int iy = iyy; iy < (ny-1); iy+=sy)
    {
        for(int ix = ixx; ix < (nx-1); ix+=sx)
        {
            double d;

            vp[iy*nx+ix] = -0.25 * (f[iy*nx+ix] -
                                    (v[nx*iy     + ix+1] + v[nx*iy     + ix-1] +
                                     v[nx*(iy+1) + ix  ] + v[nx*(iy-1) + ix  ]));

            d = fabs(vp[nx*iy+ix] - v[nx*iy+ix]);
            e[tid] = (d > e[tid]) ? d : e[tid];
        }
    }
    /*
     * Copy the points on to the original array
     */
    for (int iy = iyy; iy < (ny-1); iy+=sy)
    {
        for (int ix = ixx; ix < (nx-1); ix+=sx)
        {
            //v[nx*iy+ix] = vp[nx*iy+ix];
            w[tid] += fabs(vp[nx*iy+ix]);
        }
    }
}

/*
 * Kernel to compute the appropriate boundary weights...
 */
__global__
void weightBoundaryKernel_x(double *v, int nx, int ny, double *w, int iy)
{
    /*
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;*/
    int ixx = threadIdx.x + blockIdx.x * blockDim.x;
    int sx = blockDim.x * gridDim.x;

    /*Thread id for computation of the local weights and errors*/
    const unsigned int tid = ixx;

    if(ixx < 1)
        ixx+=sx;

    /*
     * Update the boundary points
     */
    for (int ix = ixx; ix < (nx-1); ix+=sx)
    {
        //v[nx*1      + ix] = v[nx*0     + ix];
        w[tid] += fabs(v[nx*iy + ix]);
    }
}

__global__
void weightBoundaryKernel_y(double *v, int nx, int ny, double *w, int ix)
{
    /*
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;*/
    int iyy = threadIdx.x + blockIdx.x * blockDim.x;
    int sy = blockDim.x * gridDim.x;

    /*Thread id for computation of the local weights and errors*/
    const unsigned int tid = iyy;

    if(iyy < 1)
        iyy+=sy;

    /*
     * Update the boundary points
     */
    for (int iy = iyy; iy < (ny-1); iy+=sy)
    {
        //v[nx*iy + 1]      = v[nx*iy + 0];
        w[tid] += fabs(v[nx*iy + ix]);
    }
}

__global__
void packColumn(double *v, double *col, int N, int offset)
{
    int ixx = threadIdx.x + blockIdx.x * blockDim.x;
    int sx = blockDim.x * gridDim.x;

    for(int i = ixx; i < N; i+=sx)
    {
        col[i] = v[i*offset];
    }
}

__global__
void unpackColumn(double *v, double *col, int N, int offset)
{
    int ixx = threadIdx.x + blockIdx.x * blockDim.x;
    int sx = blockDim.x * gridDim.x;

    for(int i = ixx; i < N; i+=sx)
    {
        v[i*offset] = col[i];
    }
}


/*
 * Host solver methods, which handles synchronisation and other important tasks.
 */
__host__
int solver(double *v, double *f, int nx, int ny, double eps, int nmax, struct proc_info *proc)
{
    int n = 0;

    /*Allocate memory for the secondary array vp*/
    double *vp;
    hipMalloc(&vp, nx * ny * sizeof(double));

    double *leftHalo, *rightHalo;
    hipMalloc(&leftHalo, (ny-2)* sizeof(double));
    hipMalloc(&rightHalo, (ny-2)* sizeof(double));

    /*Set the number of blocks and number of Threads for the kernel launches*/
    dim3 threadsPerBlock;
    dim3 numberOfBlocks;
    threadsPerBlock = dim3(16, 16);
    numberOfBlocks = dim3(8,8);

    /*Calculate theoretical value of total number of gpu threads*/
    const unsigned int num_gpu_threads = (numberOfBlocks.x * threadsPerBlock.x) * (numberOfBlocks.y * threadsPerBlock.y);

    /*Allocate local array for the errors and weights on device*/
    double *w_device, *e_device;
    hipMalloc(&w_device, num_gpu_threads*sizeof(double));
    hipMalloc(&e_device, num_gpu_threads*sizeof(double));

    /*Allocate memory for the resulting reduced weight and error on the device*/
    double *d_e, *d_w;
    hipMalloc(&d_w, sizeof(double));
    hipMalloc(&d_e, sizeof(double));

    /*Host weight and error*/
    double w, e = 2. * eps;

    /*Set-up for the CUB reduction*/
    void *sum_temp_storage=NULL;
    size_t sum_temp_storage_bytes = 0;
    hipcub::DeviceReduce::Sum(sum_temp_storage, sum_temp_storage_bytes, w_device, d_w, num_gpu_threads);
    hipMalloc(&sum_temp_storage,sum_temp_storage_bytes);

    void *max_temp_storage = NULL;
    size_t max_temp_storage_bytes = 0;
    hipcub::DeviceReduce::Max(max_temp_storage, max_temp_storage_bytes, e_device, d_e, num_gpu_threads);
    hipMalloc(&max_temp_storage, max_temp_storage_bytes);

    hipDeviceSynchronize();

    while ((e > eps) && (n < nmax))
    {

        jacobiStepKernel<<<numberOfBlocks, threadsPerBlock>>>(vp, v, f, nx, ny, e_device, w_device);

        /*Swap the pointer for the primary and secondary array before updating the boundary*/
        double *tmp = v;
        v = vp;
        vp = tmp;

        hipDeviceSynchronize();

        /*Communication Phase*/

        MPI_Sendrecv(&v[nx*(ny-2)+1], nx-2, MPI_DOUBLE, proc->neighbors[NORTH], 0,
                     &v[1], nx-2, MPI_DOUBLE, proc->neighbors[SOUTH], 0, proc->cartcomm, MPI_STATUS_IGNORE);

        MPI_Sendrecv(&v[nx*1 + 1], nx-2, MPI_DOUBLE, proc->neighbors[SOUTH], 0,
                     &v[nx*(ny-1) + 1], nx-2, MPI_DOUBLE, proc->neighbors[NORTH], 0, proc->cartcomm, MPI_STATUS_IGNORE);

        packColumn<<<numberOfBlocks.x*numberOfBlocks.y, threadsPerBlock.x,threadsPerBlock.y>>>(&v[nx*2 - 2], rightHalo, ny-2, nx);
        hipDeviceSynchronize();

        MPI_Sendrecv(rightHalo, ny-2, MPI_DOUBLE, proc->neighbors[EAST], 0,
                     leftHalo, ny-2,  MPI_DOUBLE, proc->neighbors[WEST], 0, proc->cartcomm, MPI_STATUS_IGNORE);

        unpackColumn<<<numberOfBlocks.x*numberOfBlocks.y, threadsPerBlock.x,threadsPerBlock.y>>>(&v[nx], leftHalo, ny-2, nx);

        packColumn<<<numberOfBlocks.x*numberOfBlocks.y, threadsPerBlock.x,threadsPerBlock.y>>>(&v[nx+1], leftHalo, ny-2, nx);
        hipDeviceSynchronize();

        MPI_Sendrecv(leftHalo, ny-2, MPI_DOUBLE, proc->neighbors[WEST], 0,
                     rightHalo, ny-2, MPI_DOUBLE, proc->neighbors[EAST], 0, proc->cartcomm, MPI_STATUS_IGNORE);

        unpackColumn<<<numberOfBlocks.x*numberOfBlocks.y, threadsPerBlock.x,threadsPerBlock.y>>>(&v[nx*2 - 1], rightHalo, ny-2, nx);

        /*End of communication phase*/

        if((n+1) % INTERVAL_ERROR_CHECK == 0)
        {
            w = 0;
            e = 0;
            /*Compute weight on the boundary*/
            if (proc->coords[0] == 0)
            {
                weightBoundaryKernel_x<<<numberOfBlocks.x*numberOfBlocks.y, threadsPerBlock.x,threadsPerBlock.y>>>(v,nx,ny,w_device,0);
            }

            if(proc->coords[0] == proc->dims[0]-1)
            {
                weightBoundaryKernel_x<<<numberOfBlocks.x*numberOfBlocks.y, threadsPerBlock.x,threadsPerBlock.y>>>(v,nx,ny,w_device,ny-1);
            }

            if(proc->coords[1] == 0)
            {
                weightBoundaryKernel_y<<<numberOfBlocks.x*numberOfBlocks.y, threadsPerBlock.x,threadsPerBlock.y>>>(v,nx,ny,w_device,0);
            }

            if(proc->coords[1] == proc->dims[1]-1)
            {
                weightBoundaryKernel_y<<<numberOfBlocks.x*numberOfBlocks.y, threadsPerBlock.x,threadsPerBlock.y>>>(v,nx,ny,w_device,nx-1);
            }

            //deviceReduce(w_device, w, num_gpu_threads);
            hipcub::DeviceReduce::Sum(sum_temp_storage, sum_temp_storage_bytes, w_device, d_w, num_gpu_threads);
            //deviceReduceMax(e_device, e, num_gpu_threads);
            hipcub::DeviceReduce::Max(max_temp_storage, max_temp_storage_bytes, e_device, d_e, num_gpu_threads);

            hipMemcpy(&e, d_e, sizeof(double), hipMemcpyDeviceToHost);
            hipMemcpy(&w, d_w, sizeof(double), hipMemcpyDeviceToHost);

            MPI_Allreduce(MPI_IN_PLACE, &e, 1, MPI_DOUBLE, MPI_MAX, proc->cartcomm);
            MPI_Allreduce(MPI_IN_PLACE, &w, 1, MPI_DOUBLE, MPI_SUM, proc->cartcomm);

            hipDeviceSynchronize();
            w /= (NX * NY);
            e /= w;
        }

        //if ((n % 10) == 0)
        //printf("%5d, %0.4e,  %0.4e\n", n, e[0], w[0]);

        n++;
    }

    /*Last "safeguard synchronisation before end of the program"*/
    hipDeviceSynchronize();

    /*Clean-up*/
    hipFree(vp);
    hipFree(w_device);
    hipFree(e_device);
    hipFree(d_e);
    hipFree(d_w);

    hipFree(leftHalo);
    hipFree(rightHalo);

    hipFree(sum_temp_storage);
    hipFree(max_temp_storage);


    if (e < eps)
        printf("Converged after %d iterations (nx=%d, ny=%d, e=%.2e)\n", n, nx, ny, e);
    else
        printf("ERROR: Failed to converge\n");

    return (e < eps ? 0 : 1);
}


